#include "hip/hip_runtime.h"
#include	<wb.h>

#define SegSize 1024

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < len)
		out[idx] = in1[idx] + in2[idx];
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;
	
	float * deviceInputA0;
	float * deviceInputA1;
	float * deviceInputB0;
	float * deviceInputB1;
	
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

	hipMalloc((void**)&deviceOutput, 2 * SegSize * sizeof(float));
	hipMalloc((void**)&deviceInputA0, SegSize * sizeof(float));
	hipMalloc((void**)&deviceInputA1, SegSize * sizeof(float));
	hipMalloc((void**)&deviceInputB0, SegSize * sizeof(float));
	hipMalloc((void**)&deviceInputB1, SegSize * sizeof(float));
	
	for (int i = 0; i < inputLength; i += SegSize * 2) {
		
		hipMemcpyAsync(deviceInputA0, hostInput1 + i, SegSize * sizeof(float),
						hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(deviceInputB0, hostInput2 + i, SegSize * sizeof(float),
						hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(deviceInputA1, hostInput1 + i + SegSize, SegSize * sizeof(float),
						hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(deviceInputB1, hostInput2 + i + SegSize, SegSize * sizeof(float),
						hipMemcpyHostToDevice, stream1);
		vecAdd <<< (SegSize - 1)/256 + 1, 256, 0, stream0 >>> (deviceInputA0, deviceInputB0, deviceOutput, SegSize);
		vecAdd <<< (SegSize - 1)/256 + 1, 256, 0, stream1 >>> (deviceInputA1, deviceInputB1, deviceOutput + SegSize, SegSize);
		hipMemcpyAsync(hostOutput + i, deviceOutput, SegSize * sizeof(float),
						hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(hostOutput + i + SegSize, deviceOutput + SegSize, SegSize * sizeof(float),
						hipMemcpyDeviceToHost, stream1);
	}

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
	hipFree(deviceInputA0);
	hipFree(deviceInputA1);
	hipFree(deviceInputB0);
	hipFree(deviceInputB1);

    return 0;
}

